#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <ctype.h>
#include <cstdio>
#include <cstdlib>
#include <math.h>
#include <time.h>
#include "cuPrintf.cu"

#define REBUILD_FREQ		5			// Rebuild after every X iterations
#define TIME_STEP 		30  			// in simulation time, in minutes
#define EXIT_COUNT		200 			// Number of iterations to do before exiting, -1 for infinite
#define FILENAME_LEN	 	256
#define ERROR 			-1 			// Generic Error val for readability
#define SECS			TIME_STEP * 60		// seconds per time step
#define GRAV_CONST	  	6.67408e-11;
#define NUM_THREADS		64
#define NUM_BLOCKS		16
#define LINE_LEN		512
#define TOL			0.05
#define GIG 			1000000000
#define MI 			1000000
#define STR_TO_DATA_T(str) strtof(str, NULL)
typedef float data_t;

int fileread_build_arrays(char* filename, data_t* h_mass, data_t* h_pos_x, data_t* h_pos_y, data_t* h_pos_z, data_t* h_vel_x, data_t* h_vel_y, data_t* h_vel_z, int len);
//__global__ void kernel_add (int arrLen, data_t* x, data_t* y, data_t* result);
__global__ void kernel_force_accum(data_t* mass, data_t* pos_x, data_t* pos_y, data_t* pos_z, data_t* vel_x, data_t* vel_y, data_t* vel_z, data_t* fma_x, data_t* fma_y, data_t* fma_z, int num_bodies, int time);

__global__ void kernel_force_zero(data_t* x, data_t* y, data_t* z, int len);

__global__ void kernel_position_update(data_t* mass, data_t* pos_x, data_t* pos_y, data_t* pos_z, data_t* vel_x, data_t* vel_y, data_t* vel_z, data_t* fma_x, data_t* fma_y, data_t* fma_z, int num_bodies, int time);

__global__ void kernel_velocity_update(data_t* mass, data_t* vel_x, data_t* vel_y, data_t* vel_z, data_t* fma_x, data_t* fma_y, data_t* fma_z, int num_bodies, int time);
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
struct timespec diff(struct timespec start, struct timespec end){
  struct timespec temp;
  if ((end.tv_nsec-start.tv_nsec)<0) {
    temp.tv_sec = end.tv_sec-start.tv_sec-1;
    temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
  } else {
    temp.tv_sec = end.tv_sec-start.tv_sec;
    temp.tv_nsec = end.tv_nsec-start.tv_nsec;
  }
  return temp;
}
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void kernel_force_accum(data_t* mass, data_t* pos_x, data_t* pos_y, data_t* pos_z, data_t* vel_x, data_t* vel_y, data_t* vel_z, data_t* fma_x, data_t* fma_y, data_t* fma_z, int num_bodies, int time){

	int idx = (blockIdx.x * blockDim.x + threadIdx.x);	
	//cuPrintf("%d\n",idx);
	if(idx < num_bodies){
	int i;
	data_t r_x; //r_y, r_z, r;
	data_t r_y;
	data_t r_z;
	data_t r;
	data_t F_part;
		for(i = 0; i < num_bodies; i++){
			if(i != idx){
				r_x = pos_x[idx] - pos_x[i];
				r_y = pos_y[idx] - pos_y[i];
				r_z = pos_z[idx] - pos_z[i];
	
				r = sqrt( (r_x* r_x) + (r_y * r_y) + (r_z * r_z) );

				// force 
				F_part = 6.67408e-11 * (mass[idx] * mass[i])/(r * r *r);

				fma_x[idx]  += F_part * r_x;
				fma_y[idx]  += F_part * r_y;
				fma_z[idx]  += F_part * r_z;
				//cuPrintf("TeST");
				//cuPrintf("%f\n",fma_x[idx]);
			}
		}
	}
}
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void kernel_force_zero(data_t* x, data_t* y, data_t* z, int len){
	int i;
int idx = (blockIdx.x * blockDim.x + threadIdx.x);
	//for(i = 0; i < len; i++)
	//{
		x[idx] = 0;
		y[idx] = 0;
		z[idx] = 0;
	//}
}
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void kernel_position_update(data_t* mass, data_t* pos_x, data_t* pos_y, data_t* pos_z, data_t* vel_x, data_t* vel_y, data_t* vel_z, data_t* fma_x, data_t* fma_y, data_t* fma_z, int num_bodies, int time){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < num_bodies){
		fma_x[i] /= mass[i];
		fma_y[i] /= mass[i];
		fma_z[i] /= mass[i];

		pos_x[i] += time * (vel_x[i] + (0.5 * fma_x[i] * time)); 
		pos_y[i] += time * (vel_y[i] + (0.5 * fma_y[i] * time)); 
		pos_z[i] += time * (vel_z[i] + (0.5 * fma_z[i] * time)); 
	}
}
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void kernel_velocity_update(data_t* mass, data_t* vel_x, data_t* vel_y, data_t* vel_z, data_t* fma_x, data_t* fma_y, data_t* fma_z, int num_bodies, int time){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < num_bodies){
		vel_x[i] += fma_x[i] * time;
		vel_y[i] += fma_y[i] * time;
		vel_z[i] += fma_z[i] * time;
	}
}
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void force_zero(data_t* x, data_t* y, data_t* z, int len){
	int i;

	for(i = 0; i < len; i++)
	{
		x[i] = 0;
		y[i] = 0;
		z[i] = 0;
	}
}
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void force_accum(data_t* mass, data_t* pos_x, data_t* pos_y, data_t* pos_z, data_t* fma_x, data_t* fma_y, data_t* fma_z, int focus, int comp){
	//  First the distance
	data_t r_x, r_y, r_z, r;

	r_x = pos_x[focus] - pos_x[comp];
	r_y = pos_y[focus] - pos_y[comp];
	r_z = pos_z[focus] - pos_z[comp];

	r = sqrt( (r_x * r_x) + (r_y * r_y) + (r_z * r_z) );

	//  then the force for the focus

	data_t F_part;

	//F_part = FORCE_PARTIAL(GRAV_CONST, mass[focus], mass[comp], r);
	F_part = 6.67408e-11 * (mass[focus] * mass[comp])/(r * r *r);

	//printf("F_part %lf | m1 %.2lf kg | m2 %.2lf kg | r %.2lf km\n", F_part, mass[focus], mass[comp], r);

	fma_x[focus]  += F_part * r_x;
	fma_y[focus]  += F_part * r_y;
	fma_z[focus]  += F_part * r_z;

	// force for the comparison
	// we know this by Newton's 3rd law

	fma_x[comp]   += -fma_x[focus];
	fma_y[comp]   += -fma_y[focus];
	fma_z[comp]   += -fma_z[focus];
}
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void position_update(data_t* mass, data_t* pos_x, data_t* pos_y, data_t* pos_z, data_t* vel_x, data_t* vel_y, data_t* vel_z, data_t* fma_x, data_t* fma_y, data_t* fma_z, int len, int time){
	//  NB, when this is invoked, fma arrays will have forces built up in force_accum()
	int i;
	for(i = 0; i < len; i++){
		// convert forces to acceleration, saves a multiply later
		fma_x[i] /= mass[i];
		fma_y[i] /= mass[i];
		fma_z[i] /= mass[i];

		pos_x[i] += time * (vel_x[i] + (0.5 * fma_x[i] * time)); 
		pos_y[i] += time * (vel_y[i] + (0.5 * fma_y[i] * time)); 
		pos_z[i] += time * (vel_z[i] + (0.5 * fma_z[i] * time)); 
		//pos_x[i] += DISPLACE(vel_x[i], fma_x[i], time);
		//pos_y[i] += DISPLACE(vel_y[i], fma_y[i], time);
		//pos_z[i] += DISPLACE(vel_z[i], fma_z[i], time);
	}
}
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void velocity_update(data_t* mass, data_t* vel_x, data_t* vel_y, data_t* vel_z, data_t* fma_x, data_t* fma_y, data_t* fma_z, int len, int time){
	// NB, when this is invoked, fma arrays should be accelerations set in position_update()
	int i;

	for(i = 0; i < len; i++){
		vel_x[i] += fma_x[i] * time;
		vel_y[i] += fma_y[i] * time;
		vel_z[i] += fma_z[i] * time;
	}
}
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
int fileread_build_arrays(char* filename, data_t* h_mass, data_t* h_pos_x, data_t* h_pos_y, data_t* h_pos_z, data_t* h_vel_x, data_t* h_vel_y, data_t* h_vel_z, int len){
	// returns true -- false
	FILE *fp = fopen(filename, "r");

	if(fp == NULL) return 0;

	int i = 0;
	char *buf = (char*) malloc(LINE_LEN);
	int buf_len = 0;
	char* tmp;

	while((i < len) && (fgets(buf, LINE_LEN - 1, fp) != NULL)){
		buf_len = strlen(buf);

		if((buf_len > 0) && (buf[buf_len - 1] == '\n'))
			buf[buf_len - 1] = '\0'; 

		// extract here
		tmp 		= strtok(buf, ",");
		h_mass[i] 	= STR_TO_DATA_T(tmp);

		tmp 		= strtok(NULL, ",");
		h_pos_x[i] 	= STR_TO_DATA_T(tmp);

		tmp 		= strtok(NULL, ",");
		h_pos_y[i] 	= STR_TO_DATA_T(tmp);

		tmp 		= strtok(NULL, ",");
		h_pos_z[i] 	= STR_TO_DATA_T(tmp);

		tmp 		= strtok(NULL, ",");
		h_vel_x[i] 	= STR_TO_DATA_T(tmp);

		tmp 		= strtok(NULL, ",");
		h_vel_y[i] 	= STR_TO_DATA_T(tmp);

		tmp 		= strtok(NULL, ",");
		h_vel_z[i] 	= STR_TO_DATA_T(tmp);

		i++;
	}
	free(buf);
	fclose(fp);
	return 1;
}
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
int main(int argc, char *argv[]){
	
	// Arrays on GPU global memory
	cudaPrintfInit();
	hipEvent_t start, stop, start1, stop1;
	float elapsed_gpu, elapsed_gpu1;

	//  grab and process the file from command line
	char* filename = (char*) malloc(sizeof(char) * FILENAME_LEN);
	int i, j, k, errCount = 0;
	int num_bodies = 0;
	int allocSize = 0;

	struct timespec diff(struct timespec start, struct timespec end);
  	struct timespec time1, time2;
  	struct timespec time_stamp;
  	int clock_gettime(clockid_t clk_id, struct timespec *tp);

	/////////////////////////device variables//////////////////////////////
	data_t *d_mass;   //mass array

	data_t *d_pos_x;  //position arrays
	data_t *d_pos_y;
	data_t *d_pos_z;

	data_t *d_vel_x;  //velocity arrays
	data_t *d_vel_y;
	data_t *d_vel_z;

	data_t *d_fma_x;  //force || acceleration arrays
	data_t *d_fma_y;
	data_t *d_fma_z;
	/////////////////////////host variables/////////////////////////////
	data_t *h_mass;   //mass array

	data_t *h_pos_x;  //position arrays
	data_t *h_pos_y;
	data_t *h_pos_z;

	data_t *h_vel_x;  //velocity arrays
	data_t *h_vel_y;
	data_t *h_vel_z;

	data_t *h_fma_x;  //force || acceleration arrays
	data_t *h_fma_y;
	data_t *h_fma_z;
	data_t *h_result;

	if(argc != 2){
		printf("\nERROR: Command line requires file name input!\n");
		exit(EXIT_FAILURE);
	}

	filename = argv[1];
	
	//  calculate number of bodies
	//  NB, the loop here is designed with only galaxy_####.csv as an expected name
	//  No error checking done for variety of inputs
	char* p = filename;
	
	while(*p){	//  Still more characters to process{
		if(isdigit(*p)){
			num_bodies *= 10;
			num_bodies += strtol(p, &p, 10);
		}
		p++;
	}

	printf("Num bodies: %d\n", num_bodies);
	// total size required for allocation
	allocSize = sizeof(data_t) * num_bodies;
	printf("Allocating memory on host\n");
	// Allocate memory on CPU
	h_mass  = (data_t*) malloc(allocSize);
	h_pos_x = (data_t*) malloc(allocSize);
	h_pos_y = (data_t*) malloc(allocSize);
	h_pos_z = (data_t*) malloc(allocSize);
	h_vel_x = (data_t*) malloc(allocSize);
	h_vel_y = (data_t*) malloc(allocSize);
	h_vel_z = (data_t*) malloc(allocSize);
	h_fma_x = (data_t*) malloc(allocSize);
	h_fma_y = (data_t*) malloc(allocSize);
	h_fma_z = (data_t*) malloc(allocSize);
	h_result = (data_t*) malloc(allocSize);
	//h_result = (data_t*) malloc(allocSize);

	if(!h_mass || !h_pos_x || !h_pos_y || !h_pos_z || !h_vel_x || !h_vel_y || !h_vel_z || !h_fma_x || !h_fma_y || !h_fma_z || !h_result){
		printf("ERROR: Array malloc issue!\n");
		return 0;
	}
	printf("Done!\n");

	
	// read the file
	printf("Reading file and building arrays...\n");
	fileread_build_arrays(filename, h_mass, h_pos_x, h_pos_y, h_pos_z, h_vel_x, h_vel_y, h_vel_z, num_bodies);
	//printf("%lf\n",h_pos_x[4]);	
	printf("Done!\n");
	//accelerations
	//force_zero(h_fma_x, h_fma_y, h_fma_z, num_bodies); 

	// Select GPU
        CUDA_SAFE_CALL(hipSetDevice(0));
	
	printf("Allocating memory on GPU...\n");	
	//Allocate memory on GPU
	CUDA_SAFE_CALL(hipMalloc((data_t**)&d_mass, allocSize));	
	CUDA_SAFE_CALL(hipMalloc((data_t**)&d_pos_x, allocSize));
	CUDA_SAFE_CALL(hipMalloc((data_t**)&d_pos_y, allocSize));
	CUDA_SAFE_CALL(hipMalloc((data_t**)&d_pos_z, allocSize));
	CUDA_SAFE_CALL(hipMalloc((data_t**)&d_vel_x, allocSize));
	CUDA_SAFE_CALL(hipMalloc((data_t**)&d_vel_y, allocSize));
	CUDA_SAFE_CALL(hipMalloc((data_t**)&d_vel_z, allocSize));
	CUDA_SAFE_CALL(hipMalloc((data_t**)&d_fma_x, allocSize));
	CUDA_SAFE_CALL(hipMalloc((data_t**)&d_fma_y, allocSize));
	CUDA_SAFE_CALL(hipMalloc((data_t**)&d_fma_z, allocSize));
	printf("Done!\n");

	// Create the cuda events
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&start1);
	hipEventCreate(&stop1);
	// Record event on the default stream
	hipEventRecord(start, 0);

	printf("Copying data on GPU...\n");
	//Transfer the data to GPU memory
	CUDA_SAFE_CALL(hipMemcpy(d_mass, h_mass, allocSize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_pos_x, h_pos_x, allocSize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_pos_y, h_pos_y, allocSize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_pos_z, h_pos_z, allocSize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_vel_x, h_vel_x, allocSize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_vel_y, h_vel_y, allocSize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_vel_z, h_vel_z, allocSize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_fma_x, h_fma_x, allocSize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_fma_y, h_fma_y, allocSize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_fma_z, h_fma_z, allocSize, hipMemcpyHostToDevice));
	printf("Done!\n");

	// Launch the kernel
    	dim3 dimBlock(NUM_THREADS, 1, 1);
	printf("dimBlock\n");
	dim3 dimGrid(NUM_BLOCKS, 1, 1);
	printf("dimGrid\n");

	//kernel_force_zero<<<dimGrid, dimBlock>>>(h_fma_x, h_fma_y, h_fma_z, num_bodies);
	hipEventRecord(start1, 0);

	for(i = 0; i < EXIT_COUNT; i++)
	{
		//printf("%d\n",i);
		kernel_force_zero<<<dimGrid, dimBlock>>>(d_fma_x, d_fma_y, d_fma_z, num_bodies);
		kernel_force_accum<<<dimGrid, dimBlock>>>(d_mass, d_pos_x, d_pos_y, d_pos_z, d_vel_x, d_vel_y, d_vel_z, d_fma_x, d_fma_y, d_fma_z, num_bodies, TIME_STEP);
		kernel_position_update<<<dimGrid, dimBlock>>>(d_mass, d_pos_x, d_pos_y, d_pos_z, d_vel_x, d_vel_y, d_vel_z, d_fma_x, d_fma_y, d_fma_z, num_bodies, TIME_STEP);
		kernel_velocity_update<<<dimGrid, dimBlock>>>(d_mass, d_vel_x, d_vel_y, d_vel_z, d_fma_x, d_fma_y, d_fma_z, num_bodies, TIME_STEP);
		
	}

	//CUDA_SAFE_CALL(hipPeekAtLastError());
	cudaPrintfDisplay(stdout, true);
  	cudaPrintfEnd();

	CUDA_SAFE_CALL(hipMemcpy(h_result, d_pos_x, allocSize, hipMemcpyDeviceToHost));

	//printf("kernel call done\n");
	hipEventRecord(stop1,0);
	hipEventSynchronize(stop1);
	hipEventElapsedTime(&elapsed_gpu1, start1, stop1);
	printf("\nGPU time for kernel execution: %lf (msec)\n", elapsed_gpu1);
	hipEventDestroy(start1);
	hipEventDestroy(stop1);
	hipDeviceSynchronize();

	// Stop and destroy the timer
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_gpu, start, stop);
	printf("\nGPU time: %lf (msec)\n", elapsed_gpu);
	hipEventDestroy(start);
	hipEventDestroy(stop);


///////////////////////////CPU turn////////////////////////////////////////////////
	printf("CPU: 'My turn son!' \n");
	printf("GPU: 'I am gonna come back tomorrow then, slow timer!' \n");

     	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time1);
	for(i = 0; i < EXIT_COUNT; i++)
	{
		//printf("Position (x, y, z) of body 5: (%f, %f, %f)\n", pos_x[4], pos_y[4], pos_z[4]);
		force_zero(h_fma_x, h_fma_y, h_fma_z, num_bodies);		

		for(j = 0; j < num_bodies; j++)
		{
			for(k = j + 1; k < num_bodies; k++)
				force_accum(h_mass, h_pos_x, h_pos_y, h_pos_z, h_fma_x, h_fma_y, h_fma_z, j, k);
		}

		position_update(h_mass, h_pos_x, h_pos_y, h_pos_z, h_vel_x, h_vel_y, h_vel_z, h_fma_x, h_fma_y, h_fma_z, num_bodies, TIME_STEP);
		velocity_update(h_mass, h_vel_x, h_vel_y, h_vel_z, h_fma_x, h_fma_y, h_fma_z, num_bodies, TIME_STEP);
		
	}

     	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time2);
     	time_stamp = diff(time1,time2);
     	printf("CPU time is %ld (msec)", (long int)(GIG * time_stamp.tv_sec + time_stamp.tv_nsec)/MI);

	int N = num_bodies;
	// Compare the results
	for(i = 0; i < N; i++) 
		{
			if (abs(h_result[i] - h_pos_x[i]) > (TOL*h_pos_x[i]))  //h_result is the output of the GPU copied to host i.e. CPU
				errCount++;
		}
	//}
		
	if (errCount > 0) {
		printf("\nERROR: TEST FAILED: %d results did not match\n", errCount);
	}
//	else if (zeroCount > 0){
//		printf("\n@ERROR: TEST FAILED: %d results (from GPU) are zero\n", zeroCount);
//	}
	else {
		printf("\nTEST PASSED: All results matched\n");
	}


}
