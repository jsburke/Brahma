
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <ctype.h>
#include <cstdio>
#include <cstdlib>
#include <math.h>
#include <time.h>

#define REBUILD_FREQ		5			// Rebuild after every X iterations
#define TIME_STEP 		30  			// in simulation time, in minutes
#define EXIT_COUNT		200 			// Number of iterations to do before exiting, -1 for infinite
#define FILENAME_LEN	 	256
#define ERROR 			-1 			// Generic Error val for readability

#define SECS			TIME_STEP * 60		// seconds per time step
#define GRAV_CONST	  	6.67408e-11;
#define NUM_THREADS		16
#define NUM_BLOCKS		1
#define LINE_LEN		512
#define EXIT_COUNT		200			//  number of iterations in loop


typedef float data_t;

////////////////////////////////////////////Function prototyping////////////////////////////////////
//__device__ void position_update(data_t* mass, data_t* pos_x, data_t* pos_y, data_t* pos_z, data_t* vel_x, data_t* vel_y, data_t* vel_z, data_t* fma_x, data_t* fma_y, data_t* fma_z, int num_bodies, int time);

//__device__ void velocity_update(data_t* mass, data_t* vel_x, data_t* vel_y, data_t* vel_z, data_t* fma_x, data_t* fma_y, data_t* fma_z, int num_bodies, int time);

__device__ void position_update(data_t mass, data_t pos_x, data_t pos_y, data_t pos_z, data_t vel_x, data_t vel_y, data_t vel_z, data_t fma_x, data_t fma_y, data_t fma_z, int num_bodies, int time);

__device__ void velocity_update(data_t mass, data_t vel_x, data_t vel_y, data_t vel_z, data_t fma_x, data_t fma_y, data_t fma_z, int num_bodies, int time);

__global__ void kernel_force_accum(data_t* mass, data_t* pos_x, data_t* pos_y, data_t* pos_z, data_t* vel_x, data_t* vel_y, data_t* vel_z, data_t* fma_x, data_t* fma_y, data_t* fma_z, int num_bodies, int time);

void force_zero(data_t* x, data_t* y, data_t* z, int len);

int fileread_build_arrays(char* filename, data_t* mass, data_t* pos_x, data_t* pos_y, data_t* pos_z, data_t* vel_x, data_t* vel_y, data_t* vel_z, int len);

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void kernel_force_accum(data_t* mass, data_t* pos_x, data_t* pos_y, data_t* pos_z, data_t* vel_x, data_t* vel_y, data_t* vel_z, data_t* fma_x, data_t* fma_y, data_t* fma_z, int num_bodies, int time)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int i;
	data_t* r_x; //r_y, r_z, r;
	data_t* r_y;
	data_t* r_z;
	data_t* r;
	data_t F_part;
	
	for(i = 0; i < num_bodies; i++){
		//int idx = index * num_bodies + i;
		if(i != idx){
			r_x[idx] = pos_x[idx] - pos_x[i];
			r_y[idx] = pos_y[idx] - pos_y[i];
			r_z[idx] = pos_z[idx] - pos_z[i];
	
			r[idx] = sqrt( (r_x[idx] * r_x[idx]) + (r_y[idx] * r_y[idx]) + (r_z[idx] * r_z[idx]) );

			// force 
			F_part = 6.67408e-11 * (mass[idx] * mass[i])/(r[idx] * r[idx] *r[idx]);

			fma_x[idx]  += F_part * r_x[idx];
			fma_y[idx]  += F_part * r_y[idx];
			fma_z[idx]  += F_part * r_z[idx];

			//fma_x[i]   += -fma_x[idx];
			//fma_y[i]   += -fma_y[idx];
			//fma_z[i]   += -fma_z[idx];
			}
		__syncthreads();
	}
	position_update(mass[idx], pos_x[idx], pos_y[idx], pos_z[idx], vel_x[idx], vel_y[idx], vel_z[idx], fma_x[idx], fma_y[idx],fma_z[idx], num_bodies, time);
	velocity_update(mass[idx], vel_x[idx], vel_y[idx], vel_z[idx], fma_x[idx], fma_y[idx], fma_z[idx], num_bodies, TIME_STEP);
	
}

__device__ void position_update(data_t mass, data_t pos_x, data_t pos_y, data_t pos_z, data_t vel_x, data_t vel_y, data_t vel_z, data_t fma_x, data_t fma_y, data_t fma_z, int num_bodies, int time)
{
	//  NB, when this is invoked, fma arrays will have forces built up in force_accum()
	//int i;
	//int index = blockIdx.x * blockDim.x + threadIdx.x;
	//int idx = blockIdx.x * blockDim.x + threadIdx.x;

	//for(i = 0; i < len; i++)
	//{
		//int idx = index * num_bodies + i;

		// convert forces to acceleration, saves a multiply later
		fma_x /= mass;
		fma_y /= mass;
		fma_z /= mass;

		pos_x += time * (vel_x + (0.5 * fma_x * time)); 
		pos_y += time * (vel_y + (0.5 * fma_y * time)); 
		pos_z += time * (vel_z + (0.5 * fma_z * time)); 
		__syncthreads();
	//}
}

__device__ void velocity_update(data_t mass, data_t vel_x, data_t vel_y, data_t vel_z, data_t fma_x, data_t fma_y, data_t fma_z, int num_bodies, int time)
{
	// NB, when this is invoked, fma arrays should be accelerations set in position_update()
	//int i;
	//int index = blockIdx.x * blockDim.x + threadIdx.x;
	//int idx = blockIdx.x * blockDim.x + threadIdx.x;

	//for(i = 0; i < num_bodies; i++)
	//{
		//int idx = index * num_bodies + i;
		vel_x += fma_x * time;
		vel_y += fma_y * time;
		vel_z += fma_z * time;
	//}
}

void	force_accum(data_t* mass, data_t* pos_x, data_t* pos_y, data_t* pos_z, data_t* fma_x, data_t* fma_y, data_t* fma_z, int focus, int comp)
{
	//  First the distance
	data_t r_x, r_y, r_z, r;

	r_x = pos_x[focus] - pos_x[comp];
	r_y = pos_y[focus] - pos_y[comp];
	r_z = pos_z[focus] - pos_z[comp];

	r = sqrt( (r_x * r_x) + (r_y * r_y) + (r_z * r_z) );

	//  then the force for the focus

	data_t F_part;

	//F_part = FORCE_PARTIAL(GRAV_CONST, mass[focus], mass[comp], r);
	F_part = 6.67408e-11 * (mass[focus] * mass[comp])/(r * r *r);

	//printf("F_part %lf | m1 %.2lf kg | m2 %.2lf kg | r %.2lf km\n", F_part, mass[focus], mass[comp], r);

	fma_x[focus]  += F_part * r_x;
	fma_y[focus]  += F_part * r_y;
	fma_z[focus]  += F_part * r_z;

	// force for the comparison
	// we know this by Newton's 3rd law

	fma_x[comp]   += -fma_x[focus];
	fma_y[comp]   += -fma_y[focus];
	fma_z[comp]   += -fma_z[focus];
}

void	position_update(data_t* mass, data_t* pos_x, data_t* pos_y, data_t* pos_z, data_t* vel_x, data_t* vel_y, data_t* vel_z, data_t* fma_x, data_t* fma_y, data_t* fma_z, int len, int time)
{
	//  NB, when this is invoked, fma arrays will have forces built up in force_accum()
	int i;

	for(i = 0; i < len; i++)
	{
		// convert forces to acceleration, saves a multiply later
		fma_x[i] /= mass[i];
		fma_y[i] /= mass[i];
		fma_z[i] /= mass[i];

		pos_x[i] += time * (vel_x[i] + (0.5 * fma_x[i] * time)); 
		pos_y[i] += time * (vel_y[i] + (0.5 * fma_y[i] * time)); 
		pos_z[i] += time * (vel_z[i] + (0.5 * fma_z[i] * time)); 
		//pos_x[i] += DISPLACE(vel_x[i], fma_x[i], time);
		//pos_y[i] += DISPLACE(vel_y[i], fma_y[i], time);
		//pos_z[i] += DISPLACE(vel_z[i], fma_z[i], time);
	}
}

void	velocity_update(data_t* mass, data_t* vel_x, data_t* vel_y, data_t* vel_z, data_t* fma_x, data_t* fma_y, data_t* fma_z, int len, int time)
{
	// NB, when this is invoked, fma arrays should be accelerations set in position_update()
	int i;

	for(i = 0; i < len; i++)
	{
		vel_x[i] += fma_x[i] * time;
		vel_y[i] += fma_y[i] * time;
		vel_z[i] += fma_z[i] * time;
	}
}


int main(int argc, char *argv[])
{
	// data_t check
	/*if(MAX_POS_POSITION == -1) //DATA_T_ERR not defined so changed it to final value of -1
	{
		printf("\nERROR: data_t not defined properly!\n");
		return 0;
	}*/ 
	//Commenting the above code because MAX_POS_POSITION is defined in octree.h which we are not including *period*

	//  grab and process the file from command line
	char*		filename = (char*) malloc(sizeof(char) * FILENAME_LEN);
	int 		i, j, k;
	int 		num_bodies = 0;
	int 		allocSize = 0;

	/////////////////////////device variables//////////////////////////////
	data_t *d_mass;   //mass array

	data_t *d_pos_x;  //position arrays
	data_t *d_pos_y;
	data_t *d_pos_z;

	data_t *d_vel_x;	 //velocity arrays
	data_t *d_vel_y;
	data_t *d_vel_z;

	data_t *d_fma_x;  //force || acceleration arrays
	data_t *d_fma_y;
	data_t *d_fma_z;
	
	/////////////////////////host variables/////////////////////////////

	data_t *h_mass;   //mass array

	data_t *h_pos_x;  //position arrays
	data_t *h_pos_y;
	data_t *h_pos_z;

	data_t *h_vel_x;	 //velocity arrays
	data_t *h_vel_y;
	data_t *h_vel_z;

	data_t *h_fma_x;  //force || acceleration arrays
	data_t *h_fma_y;
	data_t *h_fma_z;

	if(argc != 2)
	{
		printf("\nERROR: Comman line requires file name input!\n");
		exit(EXIT_FAILURE);
	}

	filename = argv[1];
	
	//  calculate number of bodies
	//  NB, the loop here is designed with only galaxy_####.csv as an expected name
	//  No error checking done for variety of inputs
	char* p = filename;
	
	while(*p)	//  Still more characters to process
	{
		if(isdigit(*p))
		{
			num_bodies *= 10;
			num_bodies += strtol(p, &p, 10);
		}
		p++;
	}

	printf("Num bodies: %d\n", num_bodies);
	
	// total size required for allocation
	allocSize = sizeof(data_t) * num_bodies;
	
	// Allocate memory on CPU
	h_mass  = (data_t*) malloc(allocSize);

	h_pos_x = (data_t*) malloc(allocSize);
	h_pos_y = (data_t*) malloc(allocSize);
	h_pos_z = (data_t*) malloc(allocSize);

	h_vel_x = (data_t*) malloc(allocSize);
	h_vel_y = (data_t*) malloc(allocSize);
	h_vel_z = (data_t*) malloc(allocSize);

	h_fma_x = (data_t*) malloc(allocSize);
	h_fma_y = (data_t*) malloc(allocSize);
	h_fma_z = (data_t*) malloc(allocSize);
	//h_result = (data_t*) malloc(allocSize);

	if(!h_mass || !h_pos_x || !h_pos_y || !h_pos_z || !h_vel_x || !h_vel_y || !h_vel_z || !h_fma_x || !h_fma_y || !h_fma_z)
	{
		printf("ERROR: Array malloc issue!\n");
		return 0;
	}


	
	// read the file
	fileread_build_arrays(filename, h_mass, h_pos_x, h_pos_y, h_pos_z, h_vel_x, h_vel_y, h_vel_z, num_bodies);

	//accelerations
	force_zero(h_fma_x, h_fma_y, h_fma_z, num_bodies); /////////////////// need to fix this ////////////////////////////

	// Select GPU
        CUDA_SAFE_CALL(hipSetDevice(0));
		
	//Allocate memory on GPU
	CUDA_SAFE_CALL(hipMalloc((data_t**)&d_mass, allocSize));	
	CUDA_SAFE_CALL(hipMalloc((data_t**)&d_pos_x, allocSize));
	CUDA_SAFE_CALL(hipMalloc((data_t**)&d_pos_y, allocSize));
	CUDA_SAFE_CALL(hipMalloc((data_t**)&d_pos_z, allocSize));
	CUDA_SAFE_CALL(hipMalloc((data_t**)&d_vel_x, allocSize));
	CUDA_SAFE_CALL(hipMalloc((data_t**)&d_vel_y, allocSize));
	CUDA_SAFE_CALL(hipMalloc((data_t**)&d_vel_z, allocSize));
	CUDA_SAFE_CALL(hipMalloc((data_t**)&d_fma_x, allocSize));
	CUDA_SAFE_CALL(hipMalloc((data_t**)&d_fma_y, allocSize));
	CUDA_SAFE_CALL(hipMalloc((data_t**)&d_fma_z, allocSize));
	//CUDA_SAFE_CALL(cudaMalloc((data_t*)&d_result, allocSize));

	if(!d_mass || !d_pos_x || !d_pos_y || !d_pos_z || !d_vel_x || !d_vel_y || !d_vel_z || !d_fma_x || !d_fma_y || !d_fma_z)
	{
		printf("ERROR: Array malloc issue in GPU!\n");
		return 0;
	}

	//Transfer the data to GPU memory
	CUDA_SAFE_CALL(hipMemcpy(d_mass, h_mass, allocSize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_pos_x, h_pos_x, allocSize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_pos_y, h_pos_y, allocSize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_pos_z, h_pos_z, allocSize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_vel_x, h_vel_x, allocSize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_vel_y, h_vel_y, allocSize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_vel_z, h_vel_z, allocSize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_fma_x, h_fma_x, allocSize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_fma_y, h_fma_y, allocSize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_fma_z, h_fma_z, allocSize, hipMemcpyHostToDevice));
	
	//free(filename);  //  file will no longer be accessed

	// Launch the kernel
    	dim3 dimBlock(NUM_THREADS, NUM_THREADS, 1);
	dim3 dimGrid(NUM_BLOCKS, NUM_BLOCKS, 1);
	
	for(i = 0; i < EXIT_COUNT; i++)
	{
		kernel_force_accum<<<dimGrid, dimBlock>>>(d_mass, d_pos_x, d_pos_y, d_pos_z, d_vel_x, d_vel_y, d_vel_z, d_fma_x, d_fma_y, d_fma_z, num_bodies, TIME_STEP);
	
	}

	// Check for errors during launch
	CUDA_SAFE_CALL(hipPeekAtLastError());
	
///////////////////////////CPU turn////////////////////////////////////////////////
	printf("CPU: 'My turn son!' \n");
	printf("GPU: 'I am gonna come tomorrow then slow timer!' \n");

	for(i = 0; i < EXIT_COUNT; i++)
	{
		//printf("Position (x, y, z) of body 5: (%f, %f, %f)\n", pos_x[4], pos_y[4], pos_z[4]);
		force_zero(h_fma_x, h_fma_y, h_fma_z, num_bodies);		

		for(j = 0; j < num_bodies; j++)
		{
			for(k = j + 1; k < num_bodies; k++)
				force_accum(h_mass, h_pos_x, h_pos_y, h_pos_z, h_fma_x, h_fma_y, h_fma_z, j, k);
		}

		position_update(h_mass, h_pos_x, h_pos_y, h_pos_z, h_vel_x, h_vel_y, h_vel_z, h_fma_x, h_fma_y, h_fma_z, num_bodies, TIME_STEP);
		velocity_update(h_mass, h_vel_x, h_vel_y, h_vel_z, h_fma_x, h_fma_y, h_fma_z, num_bodies, TIME_STEP);
		//  if we get graphics in, update screen here
	}





	
	// Free-up device and host memory
	/*CUDA_SAFE_CALL(cudaFree(d_mass));
	CUDA_SAFE_CALL(cudaFree(d_pos_x));
	CUDA_SAFE_CALL(cudaFree(d_pos_y));
	CUDA_SAFE_CALL(cudaFree(d_pos_z));
	CUDA_SAFE_CALL(cudaFree(d_vel_x));
	CUDA_SAFE_CALL(cudaFree(d_vel_y));
	CUDA_SAFE_CALL(cudaFree(d_vel_z));
	CUDA_SAFE_CALL(cudaFree(d_fma_x));
	CUDA_SAFE_CALL(cudaFree(d_fma_y));
	CUDA_SAFE_CALL(cudaFree(d_fma_z));
*/
	free(h_mass);
	free(h_pos_x);
	free(h_pos_y);
	free(h_pos_z);
	free(h_vel_x);
	free(h_vel_y);
	free(h_vel_z);
	free(h_fma_x);
	free(h_fma_y);
	free(h_fma_z);

	return 0;
}

void force_zero(data_t* x, data_t* y, data_t* z, int len)
{
	int i;

	for(i = 0; i < len; i++)
	{
		x[i] = 0;
		y[i] = 0;
		z[i] = 0;
	}
}

int fileread_build_arrays(char* filename, data_t* mass, data_t* pos_x, data_t* pos_y, data_t* pos_z, data_t* vel_x, data_t* vel_y, data_t* vel_z, int len)
{
	// returns true -- false
	FILE *fp = fopen(filename, "r");

	if(fp == NULL) return 0;

	int i = 0;
	char *buf = (char*) malloc(LINE_LEN);
	int buf_len = 0;
	char *tmp;

	while((i < len) && (fgets(buf, LINE_LEN - 1, fp) != NULL))
	{
		buf_len = strlen(buf);

		if((buf_len > 0) && (buf[buf_len - 1] == '\n'))
			buf[buf_len - 1] = '\0'; 

	tmp = strtok(buf, ",");
        tmp = strtok(NULL, ",");
        mass[i] = atof(tmp);

        tmp = strtok(NULL, ",");
        pos_x[i] = atof(tmp);

        tmp = strtok(NULL, ",");
        pos_y[i] = atof(tmp);

        tmp = strtok(NULL, ",");
        pos_z[i] = atof(tmp);

        tmp = strtok(NULL, ",");
        vel_x[i] = atof(tmp);

        tmp = strtok(NULL, ",");
        vel_y[i] = atof(tmp);

        tmp = strtok(NULL, ",");
        vel_z[i] = atof(tmp);

	i++;
	}
	free(buf);
	fclose(fp);
	return 1;
}
